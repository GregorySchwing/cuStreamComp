#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include "cuCompactor.cuh"
#include <chrono>
#include <stdlib.h>
#include <unistd.h>

using namespace std;

#define MAX_THREADS_PER_GRID (2**31)

#define THREADS_PER_WARP 32
//#define THREADS_PER_BLOCK 1024
//#define WARPS_PER_BLOCK (THREADS_PER_BLOCK/THREADS_PER_WARP)
//#define I_SIZE ((3/2)*THREADS_PER_BLOCK)

struct int_predicate
{
	__host__ __device__
	bool operator()(const int x)
	{
		return x>0;
	}
};
#define randBound (50)//<100
void initiData(int *h_data, uint NELEMENTS,uint &goodElements,bool randomOrStride){
	ushort stride = 4;
	for (int i = 0; i < NELEMENTS; ++i) {
		if(randomOrStride)
			h_data[i] = i%stride;
		else
			h_data[i] =(rand()%100 <= randBound) ? 1 : 0;
		if(h_data[i])
			goodElements++;
	}
	/*
	printf("NELE %u goodElements %u\n",NELEMENTS,goodElements);
	int sum = 0;
	int iter = 0;
	for (int i = 0; i < NELEMENTS; ++i) {
		sum += h_data[i];
		if (i%32 == 0 && i > 0){
			printf("iter %d %d\n",iter++,sum);
			sum = 0;
		}
	}
	printf("\n");
	*/
}


void printData(int *h_data, uint NELEMENTS){
	for (int i = 0; i < NELEMENTS; ++i) {
		cout<<h_data[i]<<" ";
	}
	cout<<endl;
}


void checkVector(int *h_data,uint NELEMENTS,uint NgoodElements){
	//printf("Checking: %i, %i\n",NELEMENTS,NgoodElements);
	int_predicate predicate;
	//for(int i=0;i<NgoodElements;i++){
		//printf("%d pred %d\n",h_data[i],predicate(h_data[i]));
	//}
	for(int i=0;i<NgoodElements;i++){
		assert(predicate(h_data[i]));
	}
	for(int i=NgoodElements;i<NELEMENTS;i++){
		assert(!predicate(h_data[i]));
	}
}



unsigned int NELEMENTS=0;
uint NgoodElements=0;
uint blockSize=8;


int main(){
  char outputFilename[500];
  strcpy(outputFilename, "Results.csv");
  FILE *output_file;
  if (access(outputFilename, F_OK) == 0)
  {
    // file exists
    output_file = fopen(outputFilename, "a");
  }
  else
  {
    // file doesn't exist
    output_file = fopen(outputFilename, "w");
    fprintf(output_file, "%s,%s,%s,%s\n", "Method", "NumberOfElements", "NumBlocks", "Time (ms)");
  }


srand(time(0));
	int *d_data, *d_output, *h_data;
	//data elements from 2^5 to 2^29

	// HYBRID
	for(int e=10;e<20;e++){
			NELEMENTS=(1<<10)<<e;
			NgoodElements=0;
			size_t datasize=sizeof(int)*NELEMENTS;
			//host input/output data
			h_data = (int*) malloc(datasize);
			memset(h_data,0,datasize);
			//device input data
			hipMalloc(&d_data,datasize);
			//device output data
			hipMalloc(&d_output,datasize);

			hipMemset(d_output,0,datasize);
			initiData(h_data,NELEMENTS,NgoodElements,false);

			//printData(h_data,NELEMENTS);

			hipMemcpy(d_data,h_data,datasize,hipMemcpyHostToDevice);
	//for(int e=7;e<30;e++){
		//blocksize from 32 to 1024
		// Warp method only handles blockSize 1024
		//for(int b=10;b<=10;b++){
		for(int b=5;b<=10;b++){
			//NELEMENTS=1<<e;
			// Warp method needs inputs of powers of 1024.
			blockSize=1<<b;
			//clock_t start = clock();
			int compact_length = cuCompactor::compact<int>(d_data,d_output,NELEMENTS,int_predicate(),blockSize,output_file);
			//hipDeviceSynchronize();
			//clock_t end = clock();
			//unsigned long millis = (end - start) * 1000 / CLOCKS_PER_SEC;
			assert(compact_length==NgoodElements);
			//copy back results to host
			hipMemcpy(h_data,d_output,datasize,hipMemcpyDeviceToHost);
			//printData(h_data,NELEMENTS);
			//checkVector(h_data,NELEMENTS,NgoodElements);
			hipMemset(d_output,0,datasize);
			compact_length = cuCompactor::compactHybrid<int>(d_data,d_output,NELEMENTS,int_predicate(),blockSize,output_file);
			//hipDeviceSynchronize();
			//clock_t end = clock();
			//unsigned long millis = (end - start) * 1000 / CLOCKS_PER_SEC;
			assert(compact_length==NgoodElements);
			//copy back results to host
			hipMemcpy(h_data,d_output,datasize,hipMemcpyDeviceToHost);
			//printData(h_data,NELEMENTS);
			//checkVector(h_data,NELEMENTS,NgoodElements);
		}//for blocksize
		hipMemset(d_output,0,datasize);
		int compact_length = cuCompactor::compactThrust<int>(d_data,d_output,NELEMENTS,int_predicate(),output_file);
		//hipDeviceSynchronize();
		//clock_t end = clock();
		//unsigned long millis = (end - start) * 1000 / CLOCKS_PER_SEC;
		assert(compact_length==NgoodElements);
		//copy back results to host
		hipMemcpy(h_data,d_output,datasize,hipMemcpyDeviceToHost);
		//printData(h_data,NELEMENTS);
		//checkVector(h_data,NELEMENTS,NgoodElements);


		//device memory free
		hipFree(d_data);
		hipFree(d_output);
		//host free  memory
		free(h_data);
		//printf("B,%i,%i,%i\n",NELEMENTS,blockSize,millis);
	}//for elements
	fclose(output_file);
	printf("ALL TEST PASSED");
}

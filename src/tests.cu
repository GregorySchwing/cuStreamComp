#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include "cuCompactor.cuh"
#include <chrono>
#include <stdlib.h>
using namespace std;

#define MAX_THREADS_PER_GRID (2**31)

#define THREADS_PER_WARP 32
//#define THREADS_PER_BLOCK 1024
//#define WARPS_PER_BLOCK (THREADS_PER_BLOCK/THREADS_PER_WARP)
//#define I_SIZE ((3/2)*THREADS_PER_BLOCK)

struct int_predicate
{
	__host__ __device__
	bool operator()(const int x)
	{
		return x>0;
	}
};
#define randBound (50)//<100
void initiData(int *h_data, uint NELEMENTS,uint &goodElements,bool randomOrStride){
	ushort stride = 4;
	for (int i = 0; i < NELEMENTS; ++i) {
		if(randomOrStride)
			h_data[i] = i%stride;
		else
			h_data[i] =(rand()%100 <= randBound) ? 1 : 0;
		if(h_data[i])
			goodElements++;
	}
	/*
	printf("NELE %u goodElements %u\n",NELEMENTS,goodElements);
	int sum = 0;
	int iter = 0;
	for (int i = 0; i < NELEMENTS; ++i) {
		sum += h_data[i];
		if (i%32 == 0 && i > 0){
			printf("iter %d %d\n",iter++,sum);
			sum = 0;
		}
	}
	printf("\n");
	*/
}


void printData(int *h_data, uint NELEMENTS){
	for (int i = 0; i < NELEMENTS; ++i) {
		cout<<h_data[i]<<" ";
	}
	cout<<endl;
}


void checkVector(int *h_data,uint NELEMENTS,uint NgoodElements){
	//printf("Checking: %i, %i\n",NELEMENTS,NgoodElements);
	int_predicate predicate;
	//for(int i=0;i<NgoodElements;i++){
		//printf("%d pred %d\n",h_data[i],predicate(h_data[i]));
	//}
	for(int i=0;i<NgoodElements;i++){
		assert(predicate(h_data[i]));
	}
	for(int i=NgoodElements;i<NELEMENTS;i++){
		assert(!predicate(h_data[i]));
	}
}



unsigned int NELEMENTS=0;
uint NgoodElements=0;
uint blockSize=8;


int main(){
srand(time(0));
	int *d_data, *d_output, *h_data;
	//data elements from 2^5 to 2^29
	/*
	for(int e=7;e<30;e++){
		//blocksize from 32 to 1024
		for(int b=5;b<=10;b++){
			NELEMENTS=1<<e;
			NgoodElements=0;
			blockSize=1<<b;
			size_t datasize=sizeof(int)*NELEMENTS;
			//host input/output data
			h_data = (int*) malloc(datasize);
			memset(h_data,0,datasize);
			//device input data
			hipMalloc(&d_data,datasize);
			//device output data
			hipMalloc(&d_output,datasize);

			hipMemset(d_output,0,datasize);
			initiData(h_data,NELEMENTS,NgoodElements,false);

			//printData(h_data,NELEMENTS);

			hipMemcpy(d_data,h_data,datasize,hipMemcpyHostToDevice);

			clock_t start = clock();
			cuCompactor::compact<int>(d_data,d_output,NELEMENTS,int_predicate(),blockSize);
			hipDeviceSynchronize();
			clock_t end = clock();
			unsigned long millis = (end - start) * 1000 / CLOCKS_PER_SEC;


			//copy back results to host
			hipMemcpy(h_data,d_output,datasize,hipMemcpyDeviceToHost);
			//printData(h_data,NELEMENTS);
			checkVector(h_data,NELEMENTS,NgoodElements);
			//device memory free
			hipFree(d_data);
			hipFree(d_output);
			//host free  memory
			free(h_data);
			printf("B,%i,%i,%i\n",NELEMENTS,blockSize,millis);
		}//for blocksize
	}//for elements
	*/
	// HYBRID
	/*
	for(int e=10;e<20;e++){
	//for(int e=7;e<30;e++){
		//blocksize from 32 to 1024
		// Warp method only handles blockSize 1024
		for(int b=10;b<=10;b++){
			printf("E %d\n",e);
		//for(int b=5;b<=10;b++){
			//NELEMENTS=1<<e;
			// Warp method needs inputs of powers of 1024.
			NELEMENTS=(1<<10)<<e;
			NgoodElements=0;
			blockSize=1<<b;
			size_t datasize=sizeof(int)*NELEMENTS;
			//host input/output data
			h_data = (int*) malloc(datasize);
			memset(h_data,0,datasize);
			//device input data
			hipMalloc(&d_data,datasize);
			//device output data
			hipMalloc(&d_output,datasize);

			hipMemset(d_output,0,datasize);
			initiData(h_data,NELEMENTS,NgoodElements,false);

			//printData(h_data,NELEMENTS);

			hipMemcpy(d_data,h_data,datasize,hipMemcpyHostToDevice);

			clock_t start = clock();
			int compact_length = cuCompactor::compact<int>(d_data,d_output,NELEMENTS,int_predicate(),blockSize);
			hipDeviceSynchronize();
			clock_t end = clock();
			unsigned long millis = (end - start) * 1000 / CLOCKS_PER_SEC;
			assert(compact_length==NgoodElements);
			//copy back results to host
			hipMemcpy(h_data,d_output,datasize,hipMemcpyDeviceToHost);
			//printData(h_data,NELEMENTS);
			checkVector(h_data,NELEMENTS,NgoodElements);
			//device memory free
			hipFree(d_data);
			hipFree(d_output);
			//host free  memory
			free(h_data);
			printf("B,%i,%i,%i\n",NELEMENTS,blockSize,millis);
		}//for blocksize
	}//for elements
	printf("ALL TEST PASSED");
	*/
	// HYBRID
	for(int e=10;e<20;e++){
	//for(int e=7;e<30;e++){
		//blocksize from 32 to 1024
		// Warp method only handles blockSize 1024
		for(int b=10;b<=10;b++){
		//for(int b=5;b<=10;b++){
			//NELEMENTS=1<<e;
			// Warp method needs inputs of powers of 1024.
			NELEMENTS=(1<<10)<<e;
			NgoodElements=0;
			blockSize=1<<b;
			size_t datasize=sizeof(int)*NELEMENTS;
			//host input/output data
			h_data = (int*) malloc(datasize);
			memset(h_data,0,datasize);
			//device input data
			hipMalloc(&d_data,datasize);
			//device output data
			hipMalloc(&d_output,datasize);

			hipMemset(d_output,0,datasize);
			initiData(h_data,NELEMENTS,NgoodElements,false);

			//printData(h_data,NELEMENTS);

			hipMemcpy(d_data,h_data,datasize,hipMemcpyHostToDevice);

			clock_t start = clock();
			int compact_length = cuCompactor::compactHybrid<int>(d_data,d_output,NELEMENTS,int_predicate(),blockSize);
			hipDeviceSynchronize();
			clock_t end = clock();
			unsigned long millis = (end - start) * 1000 / CLOCKS_PER_SEC;
			assert(compact_length==NgoodElements);
			//copy back results to host
			hipMemcpy(h_data,d_output,datasize,hipMemcpyDeviceToHost);
			//printData(h_data,NELEMENTS);
			checkVector(h_data,NELEMENTS,NgoodElements);
			//device memory free
			hipFree(d_data);
			hipFree(d_output);
			//host free  memory
			free(h_data);
			printf("H,%i,%i,%i\n",NELEMENTS,blockSize,millis);
		}//for blocksize
	}//for elements
	printf("ALL TEST PASSED");

}
